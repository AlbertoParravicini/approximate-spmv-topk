#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <vector>
#include <getopt.h>
#include <chrono>
#include <random>
#include <hipsparse.h> 

#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/sort.h>

#include "../common/utils/utils.hpp"
#include "../common/utils/options.hpp"
#include "../common/utils/evaluation_utils.hpp"
#include "../fpga/src/ip/coo_matrix.hpp"
#include "../fpga/src/gold_algorithms/gold_algorithms.hpp"
#include "light_spmv.cuh"

/////////////////////////////
/////////////////////////////

namespace chrono = std::chrono;
using clock_type = chrono::high_resolution_clock;

#define int_type unsigned int
#define real_type float

/////////////////////////////
/////////////////////////////

template <typename V>
__global__ void get_topk(int_type *indices, V *res, V *topk_res, int k) {
	int k_i = blockIdx.x * gridDim.x + threadIdx.x;
	if (k_i < k) {
		topk_res[k_i] = res[indices[k - 1 - k_i]];
	}
}

// cuSPARSE CSR SpMV example adapted from: https://github.com/NVIDIA/CUDALibrarySamples/blob/master/cuSPARSE/spmv_csr/spmv_csr_example.c

struct SpMV {

	int_type *ptr;
	int_type *idx;
	real_type *val;

	int_type *ptr_d;
	int_type *idx_d;
	real_type *val_d;
	half *val_d_half;

	int_type num_rows;
	int_type num_cols;
	int_type num_nnz;

	real_type *vec;
	real_type *vec_d;
	half *vec_d_half;

	real_type *res;
	real_type *res_d;
	half *res_d_half;

	int_type *row_counter;

	int num_blocks;
	int block_size_1d;

	int k;

	hipStream_t stream;
	hipsparseHandle_t handle;

	hipsparseSpMatDescr_t matrix;
    hipsparseDnVecDescr_t vec_cusparse, res_cusparse;
    void *cusparse_buffer;
    size_t buffer_size;

	// SpMV cuSPARSE coefficients, no need to change them;
    real_type alpha = 1.0f;
	real_type beta = 0.0f;
	
	// Device array with values from 0 to rows - 1, used for arg-sort to indentify rows;
	thrust::device_vector<int_type> index;
	int_type *res_topk_idx;
	real_type *res_topk_d;
	half *res_topk_d_half;

	GPU_IMPL gpu_impl;
	bool use_half_precision_gpu;

	SpMV(int_type *ptr_, int_type *idx_, real_type *val_, int_type num_rows_, int_type num_cols_, int_type num_nnz_, real_type *vec_, int k, int block_size_1d = DEFAULT_BLOCK_SIZE_1D, int debug = 0, GPU_IMPL gpu_impl=GPU_IMPL(0), bool use_half_precision_gpu=DEFAULT_USE_HALF_PRECISION_GPU) :
			ptr(ptr_), idx(idx_), val(val_), num_rows(num_rows_), num_cols(num_cols_), num_nnz(num_nnz_), vec(vec_), block_size_1d(block_size_1d), k(k), gpu_impl(GPU_IMPL(gpu_impl)), use_half_precision_gpu(use_half_precision_gpu) {
		// Compute number of blocks required in the computation;
		num_blocks = ceil(num_rows / (float) block_size_1d);
		
		// Transfer data;
		setup(debug);
	}

	void setup(int debug) {

		// Device array with values from 0 to rows - 1, used for arg-sort to indentify rows;
		index = thrust::device_vector<int_type>(num_rows);
		thrust::sequence(index.begin(), index.end());
		res_topk_idx = (int_type*) calloc(k, sizeof(int_type));

		// Setup cuSPARSE;
		hipsparseStatus_t status;
	  
		status = hipsparseCreate(&handle);
		if (status != HIPSPARSE_STATUS_SUCCESS) {
			if (debug) {
				std::cerr << "CUSPARSE Library initialisation failed" << std::endl;
				hipsparseDestroy(handle);
				exit(1);
			}
		}

		if (debug) {
			std::cout << "Create Kernel Arguments" << std::endl;
		}
		hipStreamCreate(&stream);
		hipMalloc(&ptr_d, sizeof(int_type) * (num_rows + 1));
		hipMalloc(&idx_d, sizeof(int_type) * num_nnz);
		hipMalloc(&val_d, sizeof(real_type) * num_nnz);
		hipMalloc(&vec_d, sizeof(real_type) * num_cols);
		hipMalloc(&res_d, sizeof(real_type) * num_rows);
		if (use_half_precision_gpu) {
			hipMalloc(&val_d_half, sizeof(half) * num_nnz);
			hipMalloc(&vec_d_half, sizeof(half) * num_cols);
			hipMalloc(&res_d_half, sizeof(half) * num_rows);
		}	
		res = (real_type*) calloc(num_rows, sizeof(real_type));

		// Transfer data from host to device;
		if (debug) {
			std::cout << "Write inputs into device memory" << std::endl;
		}
		hipMemcpyAsync(ptr_d, ptr, sizeof(int_type) * (num_rows + 1), hipMemcpyDefault, stream);
		hipMemcpyAsync(idx_d, idx, sizeof(int_type) * num_nnz, hipMemcpyDefault, stream);
		hipMemcpyAsync(val_d, val, sizeof(real_type) * num_nnz, hipMemcpyDefault, stream);
		hipMemcpyAsync(vec_d, vec, sizeof(real_type) * num_cols, hipMemcpyDefault, stream);
		hipMallocManaged(&row_counter, sizeof(int_type));
		hipMallocManaged(&res_topk_d, sizeof(real_type) * k);
		hipMalloc(&res_topk_d_half, sizeof(half) * k);
		row_counter[0] = 0;
		if (use_half_precision_gpu) {
			float_to_half<<<64, 1024, 0, stream>>>(val_d, val_d_half, num_nnz);
			float_to_half<<<64, 1024, 0, stream>>>(vec_d, vec_d_half, num_cols);
		}

		// Wait for data transfer on the GPU;
		hipDeviceSynchronize();
		if (use_half_precision_gpu) {
			hipsparseCreateCsr(&matrix, num_rows, num_cols, num_nnz, ptr_d, idx_d, val_d_half, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_16F);
			hipsparseCreateDnVec(&vec_cusparse, num_cols, vec_d_half, HIP_R_16F);
		} else {
			hipsparseCreateCsr(&matrix, num_rows, num_cols, num_nnz, ptr_d, idx_d, val_d, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
			hipsparseCreateDnVec(&vec_cusparse, num_cols, vec_d, HIP_R_32F);
		}
		hipsparseCreateDnVec(&res_cusparse, num_rows, res_d, HIP_R_32F);
		// Additional cuSPARSE buffer;
		hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matrix, vec_cusparse, &beta, res_cusparse, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, &buffer_size);
		hipMalloc(&cusparse_buffer, buffer_size);
	}

	float operator()(int debug) {
		if (debug) {
			std::cout << "Execute the kernel" << std::endl;
		}
		auto start = clock_type::now();

		switch(gpu_impl) {
			case CSR:
				// Use cuSPARSE;
				hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matrix, vec_cusparse, &beta, res_cusparse, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, cusparse_buffer);
				break;
			case CSR_LIGHTSPMV:
				// Use LightSpMV;
				if (use_half_precision_gpu) {
					light_spmv<<<num_blocks, block_size_1d, block_size_1d * sizeof(int_type), stream>>>(row_counter, ptr_d, idx_d, val_d_half, vec_d_half, res_d_half, num_rows);
				} else {
					light_spmv<<<num_blocks, block_size_1d, block_size_1d * sizeof(int_type), stream>>>(row_counter, ptr_d, idx_d, val_d, vec_d, res_d, num_rows);
				}
				break;
			default:
				if (debug) std::cout << "invalid spmv gpu implementation selected:" << gpu_impl << std::endl;
				exit(-1);
		}
		hipDeviceSynchronize();
		float duration_spmv_only =  chrono::duration_cast<chrono::nanoseconds>(clock_type::now() - start).count();

		// Arg-sort over results;
		if (use_half_precision_gpu && gpu_impl == CSR_LIGHTSPMV) {
			thrust::device_ptr<half> res_ptr(res_d_half);
			thrust::device_vector<half> res_vec(res_ptr, res_ptr + num_rows);
			auto key = thrust::make_permutation_iterator(thrust::make_transform_iterator(res_vec.begin(), thrust::identity<thrust::tuple<half>>{}), index.begin());
			thrust::sort_by_key(key, thrust::next(key, index.size()), index.begin());
	
			// Read back the results from the device to verify the output;
			int_type *raw_index_ptr = thrust::raw_pointer_cast(index.data() + num_rows - k);
			hipMemcpyAsync(res_topk_idx, raw_index_ptr, sizeof(int_type) * k, hipMemcpyDeviceToHost, stream);
			get_topk<<<1, 1024, 0, stream>>>(raw_index_ptr, res_d_half, res_topk_d_half, k);
			half_to_float<<<64, 1024, 0, stream>>>(res_topk_d_half, res_topk_d, k);
		} else {
			thrust::device_ptr<real_type> res_ptr(res_d);
			thrust::device_vector<real_type> res_vec(res_ptr, res_ptr + num_rows);
			auto key = thrust::make_permutation_iterator(thrust::make_transform_iterator(res_vec.begin(), thrust::identity<thrust::tuple<real_type>>{}), index.begin());
			thrust::sort_by_key(key, thrust::next(key, index.size()), index.begin());
	
			// Read back the results from the device to verify the output;
			int_type *raw_index_ptr = thrust::raw_pointer_cast(index.data() + num_rows - k);
			hipMemcpyAsync(res_topk_idx, raw_index_ptr, sizeof(int_type) * k, hipMemcpyDeviceToHost, stream);
			get_topk<<<1, 1024, 0, stream>>>(raw_index_ptr, res_d, res_topk_d, k);
		}
		
		// hipMemcpyAsync(res, res_d, sizeof(real_type) * num_rows, hipMemcpyDeviceToHost, stream);
		// Wait for computation to end;
		hipDeviceSynchronize();

		float elapsed = chrono::duration_cast<chrono::nanoseconds>(clock_type::now() - start).count();
		if (debug) {
			std::cout << "Kernel terminated" << std::endl;
			std::cout << "Computation took " << elapsed / 1e6 << " ms, spmv=" << duration_spmv_only / 1e6 << " ms, sorting=" << (elapsed - duration_spmv_only) / 1e6 << std::endl;
		}
		return duration_spmv_only;
	}

	void read_result(std::vector<real_type> &res_, std::vector<int_type> &res_idx_, int debug=0) {
		// Read output;
		for (int i = 0; i < k; i++) {
			res_idx_[i] = res_topk_idx[k - 1 - i];
			res_[i] = res_topk_d[i];
		}
	}

	long reset(real_type *vec_, int debug) {
		auto start = clock_type::now();
		vec = vec_;
		row_counter[0] = 0;

		// Fill index array;
		thrust::sequence(index.begin(), index.end());

		// Reset result vector;
		memset(res, 0, sizeof(real_type) * num_rows);

		hipMemcpy(vec_d, vec, sizeof(real_type) * num_cols, hipMemcpyDefault);
		hipMemcpy(res_d, res, sizeof(real_type) * num_rows, hipMemcpyDefault);
		if (use_half_precision_gpu) float_to_half<<<64, 1024>>>(vec_d, vec_d_half, num_cols);
		hipDeviceSynchronize();

		auto elapsed = chrono::duration_cast<chrono::nanoseconds>(clock_type::now() - start).count();
		if (debug) {
			std::cout << "Reset took " << elapsed / 1e6 << " ms" << std::endl;
		}
		return elapsed;
	}
};

/////////////////////////////
/////////////////////////////

template<typename I, typename V>
std::tuple<float, float> sw_test(coo_t<I, V> &coo, std::vector<V> &sw_res, std::vector<int_type> &res_idx_sw, std::vector<V> &res_sim_sw, V *vec, int_type top_k_value) {
	auto start_2 = clock_type::now();
	// spmv_coo_gold4(coo, sw_res.data(), vec);
	// Sort results and keep the top-K;
	// std::vector<int_type> sw_res_idx = sort_pr(sw_res.size(), sw_res.data());
	auto end_2 = clock_type::now();
	float sw_time_1 = (float) chrono::duration_cast<chrono::microseconds>(end_2 - start_2).count() / 1000;

	auto start_3 = clock_type::now();
	spmv_coo_gold_top_k(coo, vec, top_k_value, res_idx_sw.data(), res_sim_sw.data());

	// Sort the K output values;
	sort_tuples(top_k_value, res_idx_sw.data(), res_sim_sw.data());
	auto end_3 = clock_type::now();
	float sw_time_2 = (float) chrono::duration_cast<chrono::microseconds>(end_3 - start_3).count() / 1000;

	return std::make_tuple(sw_time_1, sw_time_2);
}

/////////////////////////////
/////////////////////////////

int main(int argc, char *argv[]) {

	// thrust::host_vector<float> H(4);
	// thrust::host_vector<int> I(4);
    // H[0] = 0.2;
    // H[1] = 0.1;
    // H[2] = 0.4;
	// H[3] = 0.01;
	// thrust::device_vector<float> D = H;
	// thrust::device_vector<int> index{D.size()};
	// thrust::sequence(index.begin(), index.end());
	// auto key = thrust::make_permutation_iterator(thrust::make_transform_iterator(D.cbegin(), thrust::identity<thrust::tuple<float>>{}), index.cbegin());
    // thrust::sort_by_key(key, thrust::next(key, index.size()), index.begin());
	// thrust::copy(index.cbegin(), index.cend(), I.begin());

	Options options = Options(argc, argv);
	int debug = (int) options.debug;	
	bool reset = options.reset;
	int block_size_1d = options.block_size_1d;
	int top_k_value = options.top_k_value;

	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_real_distribution<> dist(0, 1); // Uniform distribution between 0 and 1;

	int_type nnz;
	int_type rows;
	int_type cols; // Size of the dense vector multiplied by the matrix;

	std::vector<int_type> x;
	std::vector<int_type> y;
	std::vector<real_type> val_coo;

	int read_values = !options.ignore_matrix_values; // If false, all values in the matrix are = 1; Set it true only for non-graphs;
	auto start_1 = clock_type::now();
	readMtx(options.use_sample_matrix ? DEFAULT_MTX_FILE : options.matrix_path.c_str(), &x, &y, &val_coo, &rows, &cols, &nnz, 0, read_values, debug, true, false);
	// Wrap the COO matrix;
	coo_t<int_type, real_type> coo = coo_t<int_type, real_type>(x, y, val_coo);
	// Convert the COO matrix to CSR;
	int_type *ptr;
	int_type *idx;
	real_type *val;
	posix_memalign((void**) &ptr, 4096, (rows + 1) * sizeof(int_type));
	posix_memalign((void**) &idx, 4096, nnz * sizeof(int_type));
	posix_memalign((void**) &val, 4096, nnz * sizeof(real_type));
	coo2csr(ptr, idx, val, x, y, val_coo, rows, cols, false);

	// Vector multiplied by the sparse matrix;
	real_type *vec;
	posix_memalign((void**) &vec, 4096, cols * sizeof(real_type));
	create_sample_vector(vec, cols, true, true);

	// Temporary output of hardware SpMV;
	std::vector<real_type> hw_res(rows, 0);

	auto end_1 = clock_type::now();
	auto loading_time = chrono::duration_cast<chrono::milliseconds>(end_1 - start_1).count();

	if (debug) {
		std::cout << "loaded matrix with " << rows << " rows, " << cols << " columns and " << nnz << " non-zero elements" << std::endl;
		std::cout << "setup time=" << loading_time << " ms" << std::endl;
	}

   	//////////////////////////////
	// Generate software result //
	//////////////////////////////

	// Output of software SpMV, it contains all the similarities for all documents;
	std::vector<real_type> sw_res(coo.num_rows, 0);
	std::vector<real_type> res_sim_sw(top_k_value, 0);
	std::vector<int_type> res_idx_sw(top_k_value, 0);

	// Output of software SpMV, it contains all the similarities for all documents;
	std::tuple<float, float> sw_time = sw_test(coo, sw_res, res_idx_sw, res_sim_sw, vec, top_k_value);
	float sw_time_1 = std::get<0>(sw_time);
	float sw_time_2 = std::get<1>(sw_time);

	if (debug) {
		std::cout << "\nsw results =" << std::endl;
		for (int i = 0; i < top_k_value; i++) {
			std::cout << i << ") document " << res_idx_sw[i] << " = " << res_sim_sw[i] << std::endl;
		}
		std::cout << "sw errors = " << check_array_equality(sw_res.data(), res_sim_sw.data(), 10e-6, top_k_value, true) << std::endl;
		std::cout << "sw time, full matrix=" << sw_time_1 << " ms; sw time, top-k=" << sw_time_2 << " ms" << std::endl;
	}

	/////////////////////////////
	// Setup hardware ///////////
	/////////////////////////////

	auto start_4 = clock_type::now();
	SpMV spmv(ptr, idx, val, rows, cols, nnz, vec, top_k_value, block_size_1d, debug, options.gpu_impl, options.use_half_precision_gpu);
	auto end_4 = clock_type::now();
	auto gpu_setup_time = chrono::duration_cast<chrono::milliseconds>(end_4 - start_4).count();
	if (debug) {
		std::cout << "gpu setup time=" << gpu_setup_time << " ms" << std::endl;
	}

	/////////////////////////////
	// Execute the kernel ///////
	/////////////////////////////

	uint num_tests = options.num_tests;
	std::vector<float> exec_times;
	std::vector<float> readback_times;

	for (uint i = 0; i < num_tests; i++) {

		if (debug) {
			std::cout << "\nIteration " << i << ")" << std::endl;
		}
		// Create a new input vector and compute SW results;
		if (reset) {
			create_sample_vector(vec, cols, true, false, true);
			std::tuple<float, float> sw_time = sw_test(coo, sw_res, res_idx_sw, res_sim_sw, vec, top_k_value);
			sw_time_1 = std::get<0>(sw_time);
			sw_time_2 = std::get<1>(sw_time);
		}
		// Reset the computation at each iteration;
		spmv.reset(vec, debug);

		// Final output of hardware SpMV, it contains only the Top-K similarities and the Top-K indices;
		std::vector<real_type> hw_res(top_k_value);
		std::vector<int_type> hw_res_idx(top_k_value);

		auto start_5 = clock_type::now();
		// Main GPU computation;
		float spmv_only_time = spmv(debug) / 1e6;
		auto end_5 = clock_type::now();
		float gpu_exec_time = (float) chrono::duration_cast<chrono::nanoseconds>(end_5 - start_5).count() / 1e6;
		exec_times.push_back(gpu_exec_time);

		// Retrieve results;
		auto start_6 = clock_type::now();
		spmv.read_result(hw_res, hw_res_idx, debug);
		auto end_6 = clock_type::now();
		float readback_time = (float) chrono::duration_cast<chrono::nanoseconds>(end_6 - start_6).count() / 1e6;
		readback_times.push_back(readback_time);

		//////////////////////////////
		// Check correctness /////////
		//////////////////////////////
		int res_size = (int) hw_res_idx.size();
		if (debug) std::cout << "errors on indices =" << std::endl;
		int error_idx = check_array_equality(hw_res_idx.data(), res_idx_sw.data(), std::min(top_k_value, res_size), 0, debug);
		if (debug) std::cout << "errors on values =" << std::endl;
		int error = check_array_equality(hw_res.data(), res_sim_sw.data(), std::min(top_k_value, res_size), 10e-6, debug);
		if (debug) {
			std::cout << "sw results =" << std::endl;
			for (int j = 0; j < top_k_value; j++) {
				std::cout << j << ") document " << res_idx_sw[j] << " = " << res_sim_sw[j] << std::endl;
			}
			std::cout << "hw results=" << std::endl;
			for (int j = 0; j < std::min(top_k_value, res_size); j++) {
				std::cout << j << ") document " << hw_res_idx[j] << " = " << hw_res[j] << std::endl;
			}
			std::cout << "num errors on indices=" << error_idx << std::endl;
			std::cout << "num errors on values=" << error << std::endl;
			std::cout << "gpu exec time=" << gpu_exec_time << " ms" << std::endl;
		} else {
			if(i == 0) {
				std::cout << "iteration,error_idx,error_val,sw_full_time_ms,sw_topk_time_ms,hw_setup_time_ms,hw_spmv_only_time_ms,hw_exec_time_ms,readback_time_ms,k,sw_res_idx,sw_res_val,hw_res_idx,hw_res_val" << std::endl;
			}
			std::string sw_res_idx_str = "";
			std::string sw_res_val_str = "";
			std::string hw_res_idx_str = "";
			std::string hw_res_val_str = "";
			for (int j = 0; j < res_idx_sw.size(); j++) {
				sw_res_idx_str += std::to_string(res_idx_sw[j]) + ((j < res_idx_sw.size() - 1) ? ";" : "");
				sw_res_val_str += std::to_string(res_sim_sw[j]) + ((j < res_sim_sw.size() - 1) ? ";" : "");
			}
			for (int j = 0; j < hw_res_idx.size(); j++) {
				hw_res_idx_str += std::to_string(hw_res_idx[j]) + ((j < hw_res_idx.size() - 1) ? ";" : "");
				hw_res_val_str += std::to_string(hw_res[j]) + ((j < hw_res.size() - 1) ? ";" : "");
			}
			std::cout << i << "," << error_idx << "," << error << "," << sw_time_1 << "," << sw_time_2 << "," << gpu_setup_time << "," << spmv_only_time << "," << gpu_exec_time << "," << readback_time << "," << top_k_value << "," <<
					sw_res_idx_str << "," << sw_res_val_str << "," << hw_res_idx_str << "," << hw_res_val_str << std::endl;
		}
	}
	// Print summary of results;
	if (debug) {
		int old_precision = std::cout.precision();
		std::cout.precision(4);
		std::cout << "----------------" << std::endl;
		std::cout << "Mean FPGA execution time=" << mean(exec_times, 2) << "±" << st_dev(exec_times, 2) << " ms" << std::endl;
		std::cout << "Mean read-back time=" << mean(readback_times, 2) << "±" << st_dev(exec_times, 2) << " ms" << std::endl;
		std::cout << "----------------" << std::endl;
		std::cout.precision(old_precision);
	}
}
